// kernel.cu
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel function to add two vectors
__global__ void add(float *a, float *b, float *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] + b[idx];
    }
}

// Kernel function to subtract two vectors
__global__ void sub(float *a, float *b, float *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] - b[idx];
    }
}

// Kernel function to multiply two vectors
__global__ void mul(float *a, float *b, float *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] * b[idx];
    }
}

// Kernel function to divide two vectors
__global__ void div(float *a, float *b, float *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        if (b[idx] != 0) {
            result[idx] = a[idx] / b[idx];
        } else {
            printf("Warning: Division by zero at index %d\n", idx);
        }
    }
}

// Kernel function to compute the dot product of two vectors
__global__ void dot(float *a, float *b, float *result, int n) {
    __shared__ float cache[256];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0;

    while (idx < n) {
        temp += a[idx] * b[idx];
        idx += blockDim.x;
    }

    cache[threadIdx.x] = temp;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *result = cache[0];
    }
}
